#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1024

__global__ void matmul(float* A, float* B, float* C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;

    if(row < n && col < n) {
        for(int k = 0; k < n; ++k) {
            sum += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = sum;
    }
}

int main() {
    int size = N * N * sizeof(float);
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;

    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);

    // Initialize matrices
    for(int i = 0; i < N*N; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((N + blockSize.x - 1)/blockSize.x, (N + blockSize.y - 1)/blockSize.y);

    matmul<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Simple check
    printf("C[0] = %f\n", h_C[0]);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
